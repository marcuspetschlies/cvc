#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_complex.h>

#include "hlbl_kernel.cuh"


__device__ int d_coord_map ( int xi, int Li ) 
{
  return (xi >= Li / 2) ? (xi - Li) : xi;
}  // end of d_coord_map

__device__ int d_coord_map_zerohalf(int xi, int Li) 
{
  return (xi > Li / 2) ? xi - Li : ( (xi < Li / 2) ? xi : 0 );
}  // end of d_coord_map_zerohalf



__device__ __constant__ int d_gamma_permutation[16][24] = {
  {12, 13, 14, 15, 16, 17, 18, 19, 20, 21, 22, 23, 0, 1, 2, 3, 4, 5, 6, 7, 8, 9, 10, 11},
  {19, 18, 21, 20, 23, 22, 13, 12, 15, 14, 17, 16, 7, 6, 9, 8, 11, 10, 1, 0, 3, 2, 5, 4},
  {18, 19, 20, 21, 22, 23, 12, 13, 14, 15, 16, 17, 6, 7, 8, 9, 10, 11, 0, 1, 2, 3, 4, 5},
  {13, 12, 15, 14, 17, 16, 19, 18, 21, 20, 23, 22, 1, 0, 3, 2, 5, 4, 7, 6, 9, 8, 11, 10},
  {0, 1, 2, 3, 4, 5, 6, 7, 8, 9, 10, 11, 12, 13, 14, 15, 16, 17, 18, 19, 20, 21, 22, 23},
  {0, 1, 2, 3, 4, 5, 6, 7, 8, 9, 10, 11, 12, 13, 14, 15, 16, 17, 18, 19, 20, 21, 22, 23},
  {12, 13, 14, 15, 16, 17, 18, 19, 20, 21, 22, 23, 0, 1, 2, 3, 4, 5, 6, 7, 8, 9, 10, 11},
  {19, 18, 21, 20, 23, 22, 13, 12, 15, 14, 17, 16, 7, 6, 9, 8, 11, 10, 1, 0, 3, 2, 5, 4},
  {18, 19, 20, 21, 22, 23, 12, 13, 14, 15, 16, 17, 6, 7, 8, 9, 10, 11, 0, 1, 2, 3, 4, 5},
  {13, 12, 15, 14, 17, 16, 19, 18, 21, 20, 23, 22, 1, 0, 3, 2, 5, 4, 7, 6, 9, 8, 11, 10},
  {7, 6, 9, 8, 11, 10, 1, 0, 3, 2, 5, 4, 19, 18, 21, 20, 23, 22, 13, 12, 15, 14, 17, 16},
  {6, 7, 8, 9, 10, 11, 0, 1, 2, 3, 4, 5, 18, 19, 20, 21, 22, 23, 12, 13, 14, 15, 16, 17},
  {1, 0, 3, 2, 5, 4, 7, 6, 9, 8, 11, 10, 13, 12, 15, 14, 17, 16, 19, 18, 21, 20, 23, 22},
  {1, 0, 3, 2, 5, 4, 7, 6, 9, 8, 11, 10, 13, 12, 15, 14, 17, 16, 19, 18, 21, 20, 23, 22},
  {6, 7, 8, 9, 10, 11, 0, 1, 2, 3, 4, 5, 18, 19, 20, 21, 22, 23, 12, 13, 14, 15, 16, 17},
  {7, 6, 9, 8, 11, 10, 1, 0, 3, 2, 5, 4, 19, 18, 21, 20, 23, 22, 13, 12, 15, 14, 17, 16}
};
__device__ __constant__ int d_gamma_sign[16][24] = {
  {-1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
  {+1, -1, +1, -1, +1, -1, +1, -1, +1, -1, +1, -1, -1, +1, -1, +1, -1, +1, -1, +1, -1, +1, -1, +1},
  {-1, -1, -1, -1, -1, -1, +1, +1, +1, +1, +1, +1, +1, +1, +1, +1, +1, +1, -1, -1, -1, -1, -1, -1},
  {+1, -1, +1, -1, +1, -1, -1, +1, -1, +1, -1, +1, -1, +1, -1, +1, -1, +1, +1, -1, +1, -1, +1, -1},
  {+1, +1, +1, +1, +1, +1, +1, +1, +1, +1, +1, +1, +1, +1, +1, +1, +1, +1, +1, +1, +1, +1, +1, +1},
  {+1, +1, +1, +1, +1, +1, +1, +1, +1, +1, +1, +1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
  {+1, +1, +1, +1, +1, +1, +1, +1, +1, +1, +1, +1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
  {-1, +1, -1, +1, -1, +1, -1, +1, -1, +1, -1, +1, -1, +1, -1, +1, -1, +1, -1, +1, -1, +1, -1, +1},
  {+1, +1, +1, +1, +1, +1, -1, -1, -1, -1, -1, -1, +1, +1, +1, +1, +1, +1, -1, -1, -1, -1, -1, -1},
  {-1, +1, -1, +1, -1, +1, +1, -1, +1, -1, +1, -1, -1, +1, -1, +1, -1, +1, +1, -1, +1, -1, +1, -1},
  {+1, -1, +1, -1, +1, -1, +1, -1, +1, -1, +1, -1, -1, +1, -1, +1, -1, +1, -1, +1, -1, +1, -1, +1},
  {-1, -1, -1, -1, -1, -1, +1, +1, +1, +1, +1, +1, +1, +1, +1, +1, +1, +1, -1, -1, -1, -1, -1, -1},
  {+1, -1, +1, -1, +1, -1, -1, +1, -1, +1, -1, +1, -1, +1, -1, +1, -1, +1, +1, -1, +1, -1, +1, -1},
  {-1, +1, -1, +1, -1, +1, +1, -1, +1, -1, +1, -1, -1, +1, -1, +1, -1, +1, +1, -1, +1, -1, +1, -1},
  {-1, -1, -1, -1, -1, -1, +1, +1, +1, +1, +1, +1, -1, -1, -1, -1, -1, -1, +1, +1, +1, +1, +1, +1},
  {-1, +1, -1, +1, -1, +1, -1, +1, -1, +1, -1, +1, -1, +1, -1, +1, -1, +1, -1, +1, -1, +1, -1, +1}
};

__device__ inline void _d_fv_pl_eq_fv_ti_re(double* _RESTR out, const double* _RESTR in, double const re)
{
#pragma unroll
  for (int i = 0; i < 24; ++i) 
  {
    out[i] = in[i] * re;
  }
}

__device__ inline void _d_fv_eq_fv_pl_fv_ti_re(double* _RESTR out, const double* _RESTR ina, const double* _RESTR inb, double const re)
{
#pragma unroll
  for (int i = 0; i < 24; ++i) 
  {
    out[i] = ina[i] + inb[i] * re;
  }
}

__device__ inline void _d_fv_eq_gamma_ti_fv(double* _RESTR out, int gamma_index, const double* _RESTR in) 
{
#pragma unroll
  for (int i = 0; i < 24; ++i) 
  {
    out[i] = in[ d_gamma_permutation[gamma_index][i]] * d_gamma_sign[gamma_index][i];
  }
}

__device__ inline void _d_fv_ti_eq_g5(double* in_out) 
{
#pragma unroll
  for (int i = 12; i < 24; ++i) 
  {
    in_out[i] *= -1;
  }
} 

__device__ inline void _d_fv_eq_zero(double* in_out) 
{
#pragma unroll
  for (int i = 0; i < 24; ++i) 
  {
    in_out[i] = 0.0;
  }
} 

/***********************************************************
 * out has indexing

 * out[ 96 x spin-component + k ]
 * with k = 0,..., 96-1 the open index combinations of the kernel
 * spin-component = 0,..., 3
 *
 * kerv field must be pre-computed with indices
 *
 * kerv[sigma-rho][nu][lambda][mu],
 * so summation index mu is innermost
 ***********************************************************/

__device__ inline void _d_X_prepare_ev ( hipDoubleComplex * _RESTR out, const double* _RESTR in, const double *_RESTR kerv) 
{
  double sp[4][24];
  _d_fv_eq_gamma_ti_fv( sp[0], 0, in );
  _d_fv_ti_eq_g5( sp[0] );
  _d_fv_eq_gamma_ti_fv( sp[1], 1, in );
  _d_fv_ti_eq_g5( sp[1] );
  _d_fv_eq_gamma_ti_fv( sp[2], 2, in );
  _d_fv_ti_eq_g5( sp[2] );
  _d_fv_eq_gamma_ti_fv( sp[3], 3, in );
  _d_fv_ti_eq_g5( sp[3] );

#pragma unroll
  for ( int i = 0; i < 12; i++ )
  {
#pragma unroll
    for ( int k = 0; k < 96; k++ )
    {
      // sum on mu
      out[96*i+k].x = sp[0][2*i+0] * kerv[4*k+0] + sp[1][2*i+0] * kerv[4*k+1] + sp[2][2*i+0] * kerv[4*k+2] + sp[3][2*i+0] * kerv[4*k+3];

      out[96*i+k].y = sp[0][2*i+1] * kerv[4*k+0] + sp[1][2*i+1] * kerv[4*k+1] + sp[2][2*i+1] * kerv[4*k+2] + sp[3][2*i+1] * kerv[4*k+3];
    }
  }
}  // end of _d_X_prepare_ev


/***********************************************************
 * kernel for X-preparation of eigenvector
 *
 * wrapper and iterator to call _d_X_prepare_ev
 ***********************************************************/
__global__ void ker_X_prepare_ev ( hipDoubleComplex* _RESTR out, const double* _RESTR in, const double *_RESTR kerv, const int N )
{

  int const index  = blockIdx.x * blockDim.x + threadIdx.x;

  int const stride = blockDim.x * gridDim.x;

  for (int i = index; i < N; i += stride ) 
  {
    hipDoubleComplex * _out = out + 96*12 * i;
    const double * _in   = in   +  24 * i;
    const double * _kerv = kerv + 384 * i;
 
    _d_X_prepare_ev ( _out, _in, _kerv );
  }

  return;
}  // end of ker_X_prepare_ev
