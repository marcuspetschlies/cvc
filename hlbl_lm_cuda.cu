/****************************************************
 * hlbl_lm_cuda
 ****************************************************/
#include <cstdio>
#include <cstdlib>
#include <complex>
#include <cmath>
#include <functional>
#include <iostream>
#include <stdexcept>
#include <string>
#include <hip/library_types.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hipblas.h>
#include <hip/hip_complex.h>
#include <hipblas.h>

// using data_type = hipDoubleComplex;

#include "hlbl_kernel.cuh"
#include "hlbl_lm_cuda.cuh"

/***********************************************************/
/***********************************************************/

/***********************************************************
 * upload field to gpu
 ***********************************************************/
// ?? extra function ?

#if 0
/***********************************************************/
/***********************************************************/

/***********************************************************
 * compute p = V^H s
 *
 * V is nv x nx (C) = nx x nv (F)
 * s is ns x nx (C) = nx x ns (F)
 *
 * p is [nx x nv]^H x [nx x ns] = nv x ns (F) = ns x nv (C)
 *
 * HOW TO ?
 * - choose which device, hipSetDevice ?
 ***********************************************************/
int hlbl_lm_reduce ( void * const h_p, void * const h_v, void * const h_s, const int nv, const int nx, const int ns ) 
{

  hipblasHandle_t cublasH = NULL;
  hipStream_t stream = NULL;

  // like in cpu version projec(...)
  const int lda = nx;
  const int ldb = nx;
  const int ldc = nv;

  const data_type alpha = { 1.0, 0.0 };
  const data_type beta = { 0.0, 0.0 };

  data_type *d_v = nullptr;
  data_type *d_s = nullptr;
  data_type *d_p = nullptr;

  hipblasOperation_t transa = HIPBLAS_OP_C;
  hipblasOperation_t transb = HIPBLAS_OP_N;

  /* step 1: create cublas handle, bind a stream */
  CUBLAS_CHECK(hipblasCreate(&cublasH));

  CUDA_CHECK(hipStreamCreateWithFlags(&stream, hipStreamNonBlocking));
  CUBLAS_CHECK(hipblasSetStream(cublasH, stream));

  /* step 2: copy data to device */
  CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_v), sizeof(data_type) * nv * nx ) );
  CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_s), sizeof(data_type) * ns * nx ) );
  CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_p), sizeof(data_type) * ns * nv ) );

  CUDA_CHECK(hipMemcpyAsync(d_v, h_v, sizeof(data_type) * nv * nx, hipMemcpyHostToDevice, stream));

  CUDA_CHECK(hipMemcpyAsync(d_s, h_s, sizeof(data_type) * ns * nx, hipMemcpyHostToDevice, stream));

  /* step 3: compute */
  CUBLAS_CHECK( hipblasZgemm(cublasH, transa, transb, nv, ns, nx, &alpha, d_v, lda, d_s, ldb, &beta, d_p, ldc));

  /* step 4: copy data to host */
  CUDA_CHECK(hipMemcpyAsync( h_p, d_p, sizeof(data_type) * ns * nv, hipMemcpyDeviceToHost, stream));

  CUDA_CHECK(hipStreamSynchronize(stream));

  /* free resources */
  CUDA_CHECK(hipFree(d_v));
  CUDA_CHECK(hipFree(d_s));
  CUDA_CHECK(hipFree(d_p));

  CUBLAS_CHECK(hipblasDestroy(cublasH));

  CUDA_CHECK(hipStreamDestroy(stream));

  return(0);
}  // end of hlbl_lm_reduce
#endif

/***********************************************************/
/***********************************************************/

/***********************************************************
 * compute p = V^H s
 *
 * V is nv x nx (C) = nx x nv (F)
 * s is ns x nx (C) = nx x ns (F)
 *
 * p is [nx x nv]^H x [nx x ns] = nv x ns (F) = ns x nv (C)
 *
 ***********************************************************/
int hlbl_lm_reduce ( hipStream_t stream, hipblasHandle_t cublasH, double _Complex * const h_p, 
    cuda_data_type * const d_v, cuda_data_type * const d_s, const int nv, const int nx, const int ns ) 
{
  cuda_data_type *d_p = nullptr;

  // like in cpu version projec(...)
  const int lda = nx;
  const int ldb = nx;
  const int ldc = nv;

  const cuda_data_type alpha = { 1.0, 0.0 };
  const cuda_data_type beta  = { 0.0, 0.0 };

  hipblasOperation_t transa = HIPBLAS_OP_C;
  hipblasOperation_t transb = HIPBLAS_OP_N;

  /* copy data to device */
  CUDA_CHECK_MALLOC(hipMalloc(reinterpret_cast<void **>(&d_p), sizeof(cuda_data_type) * ns * nv ) );

  /* linear algebra computation */
  CUBLAS_CHECK( hipblasZgemm(cublasH, transa, transb, nv, ns, nx, &alpha, d_v, lda, d_s, ldb, &beta, d_p, ldc));

  /* step 4: copy data to host */
  CUDA_CHECK(hipMemcpyAsync( h_p, d_p, sizeof(cuda_data_type) * ns * nv, hipMemcpyDeviceToHost, stream));

  CUDA_CHECK(hipStreamSynchronize(stream));

  /* free resources */
  CUDA_CHECK(hipFree(d_p));

  return(0);
}  // end of hlbl_lm_reduce

/***********************************************************
 * compute p = V^H x S^T
 *
 * V is nv x nx (C) = nx x nv (F)
 * S is nx x ns (C) = ns x nx (F) NOT TRUE ANYMORE
 *
 * p is   [nx x nv]^H x [ns x nx]^T  = [nv x nx]^* x [nx x ns]
 *      = [nv x ns] (F)
 *      = [ns x nv] (C)
 *
 * NOTE: nx is the number of sites!
 *       NOT the number of fermion field components
 ***********************************************************/
int project_v_dag_g_v ( hipStream_t stream, hipblasHandle_t cublasH, double _Complex * const h_p, 
    const double * d_v, const double * kervx, const int nv, const int nx ) 
{
  dim3 blockSize(CUDA_BLOCK_SIZE);
  dim3 gridSize( ( (nx + CUDA_BLOCK_SIZE - 1 ) / CUDA_BLOCK_SIZE) );

  hipDoubleComplex * d_p = nullptr;
  hipDoubleComplex * d_s = nullptr;
  
  /* number of components; here for X */
  const int ns = 96;

  // like in cpu version project(...)
  const int lda = 12*nx;
  const int ldb = ns;
  const int ldc = nv;

  /* C <- alpha A x B + beta C */
  const hipDoubleComplex alpha = { 1.0, 0.0 };
  const hipDoubleComplex beta  = { 0.0, 0.0 };

  /* reshaping of matrices */
  hipblasOperation_t transa = HIPBLAS_OP_C;
  hipblasOperation_t transb = HIPBLAS_OP_T;

  /* device memory for projection coefficients */
  CUDA_CHECK_MALLOC (hipMalloc(reinterpret_cast<void **>(&d_p), sizeof(hipDoubleComplex) * ns * nv ) );

  /* device memory for s
   * s is a fermion field with V x 12 [spin-color] x 96 [kernel components] */
  CUDA_CHECK_MALLOC ( hipMalloc(reinterpret_cast<void **>(&d_s), sizeof(hipDoubleComplex) * ns * 12 * nx ) );

  /* loop on vectors */
  for ( int iv = 0; iv < nv; iv++ )
  {
#if 0
    /* prepare s, i.e. apply vertex 
     * kernel call, add parallelization info to call */
    ker_X_prepare_ev<<< gridSize, blockSize >>>( d_s, d_v + iv*24*nx, kervx, nx );
#endif  // if 0

    /* linear algebra computation */
    CUBLAS_CHECK( hipblasZgemm(cublasH, transa, transb, nv, ns, 12*nx, &alpha, reinterpret_cast<const hipDoubleComplex *>(d_v), lda, d_s, ldb, &beta, d_p, ldc));

    /* copy data to host */
    CUDA_CHECK(hipMemcpyAsync( h_p + iv * ns*nv, d_p, sizeof(hipDoubleComplex) * ns * nv, hipMemcpyDeviceToHost, stream));

  }  /* end of loop on eigenvectors */

  CUDA_CHECK(hipStreamSynchronize(stream));

  /* free resources */
  CUDA_CHECK(hipFree(d_p));
  CUDA_CHECK(hipFree(d_s));

  return(0);
}  // end of project_v_dag_g_v
