/****************************************************
 * hlbl_lm_cuda
 ****************************************************/
#include <cstdio>
#include <cstdlib>
#include <complex>
#include <cmath>
#include <functional>
#include <iostream>
#include <stdexcept>
#include <string>
#include <hip/library_types.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hipblas.h>
#include <hip/hip_complex.h>
#include <hipblas.h>

// using data_type = hipDoubleComplex;

#include "hlbl_kernel.cuh"
#include "hlbl_lm_cuda.cuh"

namespace cvc {

/***********************************************************/
/***********************************************************/

/***********************************************************
 * compute p = V^H s
 *
 * V is nv x nx (C) = nx x nv (F)
 * s is ns x nx (C) = nx x ns (F)
 *
 * p is [nx x nv]^H x [nx x ns] = nv x ns (F) = ns x nv (C)
 *
 ***********************************************************/
int hlbl_lm_reduce ( hipStream_t stream, hipblasHandle_t cublasH, double _Complex * const h_p, 
    cuda_data_type * const d_v, cuda_data_type * const d_s, const int nv, const int nx, const int ns ) 
{
  cuda_data_type *d_p = nullptr;

  // like in cpu version projec(...)
  const int lda = nx;
  const int ldb = nx;
  const int ldc = nv;

  const cuda_data_type alpha = { 1.0, 0.0 };
  const cuda_data_type beta  = { 0.0, 0.0 };

  hipblasOperation_t transa = HIPBLAS_OP_C;
  hipblasOperation_t transb = HIPBLAS_OP_N;

  /* copy data to device */
  CUDA_CHECK_MALLOC(hipMalloc(reinterpret_cast<void **>(&d_p), sizeof(cuda_data_type) * ns * nv ) );

  /* linear algebra computation */
  CUBLAS_CHECK( hipblasZgemm(cublasH, transa, transb, nv, ns, nx, &alpha, d_v, lda, d_s, ldb, &beta, d_p, ldc));

  /* step 4: copy data to host */
  CUDA_CHECK(hipMemcpyAsync( h_p, d_p, sizeof(cuda_data_type) * ns * nv, hipMemcpyDeviceToHost, stream));

  CUDA_CHECK(hipStreamSynchronize(stream));

  /* free resources */
  CUDA_CHECK(hipFree(d_p));

  return(0);
}  // end of hlbl_lm_reduce

/***********************************************************/
/***********************************************************/


/***********************************************************
 * compute p = V^H x S^T
 *
 * V is nv x nx (C) = nx x nv (F)
 * S is nx x ns (C) = ns x nx (F) 
 *
 * p is   [nx x nv]^H x [ns x nx]^T  = [nv x nx]^* x [nx x ns]
 *      = [nv x ns] (F)
 *      = [ns x nv] (C)
 *
 * NOTE: nx is the number of sites!
 *       NOT the number of fermion field components
 ***********************************************************/
int project_v_dag_g_v ( hipStream_t stream, hipblasHandle_t cublasH, double _Complex * const h_p, 
    const double * d_v, const double * kervx, const int nv, const int nx ) 
{
  dim3 blockSize(CUDA_BLOCK_SIZE);
  dim3 gridSize( ( (nx + CUDA_BLOCK_SIZE - 1 ) / CUDA_BLOCK_SIZE) );

  hipDoubleComplex * d_p = nullptr;
  hipDoubleComplex * d_s = nullptr;
  
  /* number of components; here for X */
  const int ns = 96;

  // like in cpu version project(...)
  const int lda = 12*nx;
  const int ldb = ns;
  const int ldc = nv;
  //const int lda = ns;
  //const int ldb = 12*nx;
  //const int ldc = 12*nx;

  /* C <- alpha A x B + beta C */
  const hipDoubleComplex alpha = { 1.0, 0.0 };
  const hipDoubleComplex beta  = { 0.0, 0.0 };

  /* reshaping of matrices */
  hipblasOperation_t transa = HIPBLAS_OP_C;
  hipblasOperation_t transb = HIPBLAS_OP_T;
  //hipblasOperation_t transa = HIPBLAS_OP_N;
  //hipblasOperation_t transb = HIPBLAS_OP_N;

  /* device memory for projection coefficients */
  size_t bytes = ( sizeof(hipDoubleComplex) * ns ) * nv;
  CUDA_CHECK_MALLOC (hipMalloc(reinterpret_cast<void **>(&d_p), bytes ) );

  /* device memory for s
   * s is a fermion field with V x 12 [spin-color] x 96 [kernel components] */
  bytes = ( sizeof(hipDoubleComplex) * ns ) * 12 * nx;
  CUDA_CHECK_MALLOC ( hipMalloc(reinterpret_cast<void **>(&d_s), bytes ) );

  /* loop on vectors */
  for ( size_t iv = 0; iv < nv; iv++ )
  {
    size_t offset = iv * 24 * nx;

    const double * _d_v = d_v + offset;

    /* prepare s, i.e. apply vertex 
     * kernel call, add parallelization info to call */
    ker_X_prepare_ev<<< gridSize, blockSize, 0, stream >>>( d_s, _d_v, kervx, nx );
    
    /* linear algebra computation 
     * 
     * d_v is nv   x 12nx (C) = 12nx x nv   (F)
     * d_s is 12nx x ns   (C) = ns   x 12nx (F)
     */
    //                                 C       T       m      n    k
    CUBLAS_CHECK( hipblasZgemm(cublasH, transa, transb, nv, ns, 12*nx, &alpha, reinterpret_cast<const hipDoubleComplex *>(d_v), lda, d_s, ldb, &beta, d_p, ldc));
    
    /* copy data to host */
    offset = iv * ns * nv;
    bytes = sizeof(hipDoubleComplex) * ns * nv;

    CUDA_CHECK_MEMCPY ( hipMemcpyAsync( h_p + offset, d_p, bytes, hipMemcpyDeviceToHost, stream));

    CUDA_CHECK(hipStreamSynchronize(stream));

  }  /* end of loop on eigenvectors */

  /* free resources */
  CUDA_CHECK_FREE(hipFree(d_p));
  CUDA_CHECK_FREE(hipFree(d_s));

  return(0);
}  // end of project_v_dag_g_v


/***********************************************************
 *
 ***********************************************************/
// int apply_kernel ( hipStream_t stream, hipblasHandle_t cublasH, double * d_out, const double * d_in, const double * kervx,  const int nx ) 
int apply_kernel ( hipStream_t stream, hipblasHandle_t cublasH, hipDoubleComplex * d_out, const double * d_in, const double * kervx,  const int nx ) 
{
  dim3 blockSize(CUDA_BLOCK_SIZE);
  dim3 gridSize( ( (nx + CUDA_BLOCK_SIZE - 1 ) / CUDA_BLOCK_SIZE) );
  
  /* prepare s, i.e. apply vertex 
   * kernel call, add parallelization info to call */
  test_kernel<<< gridSize, blockSize >>>( d_out, d_in, kervx, nx );

  CUDA_CHECK(hipStreamSynchronize(stream));

  /* free resources */

  return(0);
}  // end of apply_kernel

}
